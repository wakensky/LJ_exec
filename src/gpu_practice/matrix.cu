
#include <hip/hip_runtime.h>
#include<iostream>
#include<random>

using namespace std;

__global__ void matrix_mul(float* a, float* b, float* c, int m, int n, int k){
  //计算行和列
  const int row = blockIdx.y * blockDim.y + threadIdx.y;
  const int column = blockIdx.x * blockDim.x + threadIdx.x;
  if(row < m && column < k){
    int temp = 0;
    //每一线程负责一次的向量计算
    for(int i = 0; i < n; i++){
      temp += a[row * n + i] * b[i * k + column];
    }
    c[row * k + column] = temp;
  }
}

void dot(float* a, float* b, float* c, int m, int n, int k){
  //在gpu上分配内存
  float* dev_a;
  float* dev_b;
  float* dev_c;
  hipMalloc((void**)&dev_a, m*n*sizeof(float));
  hipMalloc((void**)&dev_b, n*k*sizeof(float));
  hipMalloc((void**)&dev_c, m*k*sizeof(float));
  //copy
  hipMemcpy(dev_a, a, m*n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, n*k*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_c, c, m*k*sizeof(float), hipMemcpyHostToDevice);
  //设置线程
  const int thread_size = 32;
  const int row_block_size = (m + thread_size - 1) / thread_size;
  const int column_block_size = (k + thread_size - 1) / thread_size;
  dim3 blocks(thread_size, thread_size);
  dim3 grids(column_block_size, row_block_size);
  //进入kernel
  matrix_mul<<<grids,blocks>>>(dev_a, dev_b, dev_c, m, n, k);
  //拷贝dev_c到host端
  hipMemcpy(c, dev_c, m*k*sizeof(float), hipMemcpyDeviceToHost);
  cout << "矩阵乘法计算完成" << endl;
}

void init_data(float* matrix, int size){
  default_random_engine random;
  uniform_int_distribution<int> range(0,10.0);
  for(int i = 0; i < size; i++){
    matrix[i] = (float)range(random);
  }
}

void cpu_dot(float* a, float* b, float* c, int m , int n, int k){
  for(int i = 0; i < m; i++){
    for(int j = 0; j < k; j++){
      int temp = 0;
      for(int l = 0; l < n; l++){
        temp += a[i * n + l] * b[l * k + j];
      }
      c[i * k + j] = temp;
    }
  }
}

void print_matrix(float* matrix, int m, int n){
  for(int i = 0; i < m; i++){
    for(int j = 0; j < n; j++){
      cout << matrix[i * n + j] << "  ";
    }
    cout << endl;
  }
}

int main(){
  int m = 10, n = 3, k = 3;
  //有两个host matrix
  float host_a[m*n];
  float host_b[n*k];
  float host_c[m*k];
  float answer[m*k];
  //初始化数据
  init_data(host_a, m*n);
  init_data(host_b, n*k);
  cout << "初始化完成" << endl;
  print_matrix(host_a, m,n);
  print_matrix(host_b, n,k);
  //给dot函数计算矩阵相乘, m, n, k为矩阵size,结果存放在host_c
  dot(host_a, host_b, host_c, m, n, k);
  cout << "kernel完成" << endl;
  cpu_dot(host_a, host_b, answer, m, n, k);
  cout << "cpu完成" << endl;
  return 0;
}