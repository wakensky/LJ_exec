#include "hip/hip_runtime.h"
#include"scan.cuh"
#include"hip/hip_runtime.h"

#include<chrono>
#include<random>
#include<iostream>
#include<math.h>

using namespace std;
using namespace chrono;

int main(){
  for(int i = 5; i <= 28; i++){
    size_t size = pow(2,i);
    int* input = new int[size];
    int* gpu_output = new int[size];
    int* cpu_output = new int[size];
    default_random_engine random;
    uniform_int_distribution<> range(-5,5);
    for(size_t i = 0; i < size; i++){
      input[i] = range(random);
    }
    time_point<system_clock> start, end;
    start = system_clock::now();
    gpu_prefix_scan(input, gpu_output, size);
    hipDeviceSynchronize();
    end = system_clock::now();
    float gpu_time = duration_cast<microseconds>(end - start).count();
    cout << "gpu: " << i << " " << gpu_time/1000 << " ms" << endl;

    start = system_clock::now();
    cpu_prefix_sum(input, cpu_output, size);
    end = system_clock::now();
    float cpu_time = duration_cast<microseconds>(end - start).count();
    cout << "cpu: " << i << " " << cpu_time/1000 << " ms" << endl;
    cout << endl;
    check(gpu_output, cpu_output, size);
    delete [] input;
    delete [] gpu_output;
    delete [] cpu_output;
  }
  return 0;
}

