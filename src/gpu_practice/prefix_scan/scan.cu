#include"scan.cuh"
#include"kernel.cuh"

#include<iostream>

using namespace std;

//用户只需要传进来主机的input和output数组
void gpu_prefix_scan(const int* input, int* output, size_t size){
  //block_sum为每个block的reduction结果,因此需要先设定block和grid
  //只需要一维的grid和一维的block
  const int threads_per_block = 512;
  //每个线程处理两个数据因此
  const int elements_per_block = 2 * threads_per_block;
  const size_t num_blocks = (size + elements_per_block - 1) / elements_per_block;
  //set dim3
  dim3 blocks(threads_per_block);
  dim3 grids(num_blocks);


  int* block_sum = new int[num_blocks+1];
  //device data
  int* dev_input_data;
  int* dev_output_data;
  int* dev_block_sum;

  //分配内存
  const size_t data_bytes = size * sizeof(int);

  hipMalloc((void**)&dev_input_data, data_bytes);
  hipMalloc((void**)&dev_output_data, data_bytes);
  hipMalloc((void**)&dev_block_sum, num_blocks*sizeof(int));

  //copy data to device
  hipMemcpy(dev_input_data, input, data_bytes, hipMemcpyHostToDevice);
  //launch kernel to compute every block prefix sum
  prefix_sum<<<grids, blocks,elements_per_block*sizeof(int)>>>(dev_input_data, dev_output_data, dev_block_sum, elements_per_block);
  hipMemcpy(block_sum, dev_block_sum, num_blocks*sizeof(int), hipMemcpyDeviceToHost);
  block_scan(block_sum, num_blocks);
  hipMemcpy(dev_block_sum, block_sum, num_blocks*sizeof(int), hipMemcpyHostToDevice);

  //对每个元素加上对应前一个block的和
  //add<<<grids,elements_per_block>>>(dev_output_data, dev_block_sum,dev_input_data);
  add<<<grids,elements_per_block,elements_per_block*sizeof(int)>>>(dev_output_data, dev_block_sum,dev_input_data);
  //copy data from device
  hipMemcpy(output, dev_output_data, data_bytes, hipMemcpyDeviceToHost);
  hipFree(dev_input_data);
  hipFree(dev_output_data);
  hipFree(dev_block_sum);
  delete [] block_sum;
}

void cpu_prefix_sum(int* input, int* output, size_t n){
  output[0] = input[0];
  for(size_t i = 1; i < n; i++){
    output[i] = output[i-1] + input[i];
  }
}

void check(int* answer, int* test, size_t size){
  for(size_t i = 0; i < size; i++){
    if(answer[i] != test[i]){
      cout << "计算失败" << endl;
      return;
    }
  }
  cout << "计算成功" << endl;
}