#include "hip/hip_runtime.h"
#include"kernel.cuh"

__global__ void prefix_sum(int *input, int *output, int *block_sum, size_t data_per_block){
  //copy correspoding data from global memory to shared memory
  //block中threadIdx.x的范围在[0,data_per_block/2 - 1]
  const size_t data_offset = blockIdx.x *data_per_block;
  extern __shared__ int block_data[];
  block_data[2*threadIdx.x] = input[data_offset + 2*threadIdx.x];
  block_data[2*threadIdx.x + 1] = input[data_offset + 2*threadIdx.x + 1];
  //make sure all data is ready
  __syncthreads();
  //up-sweep
  //step = [0~log(data_per_block)-1]
  //offset控制步伐
  int offset = 1;
  for(int d = data_per_block >> 1; d > 0; d >>= 1){
    //在读写之前设置barrier
    __syncthreads();
    if(threadIdx.x < d){
      int a = (2*threadIdx.x+1)*offset - 1;
      int b = (2*threadIdx.x+2)*offset - 1;
      block_data[b] += block_data[a];
    }
    offset <<= 1;
  }
  __syncthreads();
  //进入down-sweep之前需要保存block sum和清0,交给第一个线程去做
  if(0 == threadIdx.x){
    block_sum[blockIdx.x+1] = block_data[data_per_block - 1];
    block_data[data_per_block - 1] = 0;
  }

  //进入down_sweep;
  for(int d = 1; d < data_per_block; d <<= 1){
    offset >>= 1;
    __syncthreads();
    if(threadIdx.x <  d){
      int a = (2*threadIdx.x + 1)*offset - 1;
      int b = (2*threadIdx.x + 2)*offset - 1;
      int temp = block_data[b];
      block_data[b] += block_data[a];
      block_data[a] = temp;
    }
  }
  __syncthreads();

  //copy shared memory to global memory
  output[data_offset + 2*threadIdx.x] = block_data[2*threadIdx.x];
  output[data_offset + 2*threadIdx.x + 1] = block_data[2*threadIdx.x + 1];
}

/*__global__ void add(int *output, const int *block_sum, const int* input){
  output[blockDim.x * blockIdx.x + threadIdx.x] += input[blockDim.x * blockIdx.x + threadIdx.x];
  __syncthreads();
  if(blockIdx.x >= 1){
    output[blockDim.x * blockIdx.x + threadIdx.x] += block_sum[blockIdx.x - 1];
  }
}*/

__global__ void add(int *output, const int *block_sum, const int* input){
  extern __shared__ int out [];
  out[threadIdx.x] = output[blockDim.x * blockIdx.x + threadIdx.x];
  __syncthreads();
  out[threadIdx.x] += input[blockDim.x * blockIdx.x + threadIdx.x] + block_sum[blockIdx.x];
  __syncthreads();
  output[blockDim.x * blockIdx.x + threadIdx.x] = out[threadIdx.x];
}

void block_scan(int* block_sum, size_t num_blocks){
  for(size_t i = 1; i < num_blocks; i++){
    block_sum[i] += block_sum[i-1];
  }
}