#include "hip/hip_runtime.h"
//suppose 有n = 1 << 20 int data to do prefix sum(exclusive)
//processors is less than data
//1024 blocks, 512 threads/block, every thread can handle 2 int
//then, look at how kernel work

#include<random>
#include<iostream>
#include<cstdlib>
#include<cstdio>

using namespace std;

__global__ void prefix_sum(int *input, int *output, int *block_sum, int data_per_block){
  //copy correspoding data from global memory to shared memory
  //block中threadIdx.x的范围在[0,data_per_block/2 - 1]
  const int data_offset = blockIdx.x *data_per_block;
  extern __shared__ int block_data[];
  block_data[2*threadIdx.x] = input[data_offset + 2*threadIdx.x];
  block_data[2*threadIdx.x + 1] = input[data_offset + 2*threadIdx.x + 1];
  //make sure all data is ready
  __syncthreads();
  //up-sweep
  //step = [0~log(data_per_block)-1]
  //offset控制步伐
  int offset = 1;
  for(int d = data_per_block >> 1; d > 0; d >>= 1){
    //在读写之前设置barrier
    __syncthreads();
    if(threadIdx.x < d){
      int a = (2*threadIdx.x+1)*offset - 1;
      int b = (2*threadIdx.x+2)*offset - 1;
      block_data[b] += block_data[a];
    }
    offset <<= 1;
  }
  __syncthreads();
  //进入down-sweep之前需要保存block sum和清0,交给第一个线程去做
  if(0 == threadIdx.x){
    block_sum[blockIdx.x] = block_data[data_per_block - 1];
    block_data[data_per_block - 1] = 0;
  }

  __syncthreads();

  //进入down_sweep;
  for(int d = 1; d < data_per_block; d <<= 1){
    offset >>= 1;
    __syncthreads();
    if(threadIdx.x <  d){
      int a = (2*threadIdx.x + 1)*offset - 1;
      int b = (2*threadIdx.x + 2)*offset - 1;
      int temp = block_data[b];
      block_data[b] += block_data[a];
      block_data[a] = temp;
    }
  }
  __syncthreads();

  //copy shared memory to global memory
  output[data_offset + 2*threadIdx.x] = block_data[2*threadIdx.x];
  output[data_offset + 2*threadIdx.x + 1] = block_data[2*threadIdx.x + 1];
}

__global__ void add(int *output, const int *block_sum, const int* input){
  output[blockDim.x * blockIdx.x + threadIdx.x] += input[blockDim.x * blockIdx.x + threadIdx.x];
  __syncthreads();
  if(blockIdx.x >= 1){
    output[blockDim.x * blockIdx.x + threadIdx.x] += block_sum[blockIdx.x - 1];
  }
}

void cpu_prefix_sum(int* input, int* output, int n){
  output[0] = input[0];
  for(int i = 1; i < n; i++){
    output[i] = output[i-1] + input[i];
  }
}

void check(int* answer, int* test, int size){
  for(int i = 0; i < size; i++){
    if(answer[i] != test[i]){
      cout << "计算失败" << endl;
      return;
    }
  }
  cout << "计算成功" << endl;
}

const int size = 1 << 28;
//block_sum为每个block的reduction结果,因此需要先设定block和grid
//只需要一维的grid和一维的block
const int threads_per_block = 512;
//每个线程处理两个数据因此
const int elements_per_block = 2 * threads_per_block;
const int num_blocks = (size + elements_per_block - 1) / elements_per_block;

int* input_data = new int[size];
int* output_data = new int[size];
int* block_sum = new int[num_blocks];
int* cpu_output = new int[size];
int main(int argc, char** argv){
  cout << size <<endl;
  //generate input in host
  //填充
  default_random_engine random;
  uniform_int_distribution<signed> range(-10,10);
  for(int i = 0; i < size; i++){
    input_data[i] = range(random);
  }

  //device data
  int* dev_input_data;
  int* dev_output_data;
  int* dev_block_sum;

  //分配内存
  const size_t data_bytes = size * sizeof(int);

  hipMalloc((void**)&dev_input_data, data_bytes);
  hipMalloc((void**)&dev_output_data, data_bytes);

  //set dim3
  dim3 blocks(threads_per_block);
  dim3 grids(num_blocks);

  hipMalloc((void**)&dev_block_sum, num_blocks*sizeof(int));

  //copy data to device
  hipMemcpy(dev_input_data, input_data, data_bytes, hipMemcpyHostToDevice);
  //launch kernel to compute every block prefix sum
  prefix_sum<<<grids, blocks,elements_per_block*sizeof(int)>>>(dev_input_data, dev_output_data, dev_block_sum, elements_per_block);

  hipMemcpy(block_sum, dev_block_sum, num_blocks*sizeof(int), hipMemcpyDeviceToHost);
  //串行作一次求和
  for(int i = 1; i < num_blocks; i++){
    block_sum[i] += block_sum[i-1];
  }
  hipMemcpy(dev_block_sum, block_sum, num_blocks*sizeof(int), hipMemcpyHostToDevice);

  //对每个元素加上对应前一个block的和
  add<<<grids,elements_per_block>>>(dev_output_data, dev_block_sum,dev_input_data);
  hipDeviceSynchronize();
  //copy data from device
  hipMemcpy(output_data, dev_output_data, data_bytes, hipMemcpyDeviceToHost);

  hipFree(dev_input_data);
  hipFree(dev_output_data);
  hipFree(dev_block_sum);
  
  //串行计算
  cpu_prefix_sum(input_data, cpu_output, size);
  check(cpu_output, output_data, size);
  delete [] input_data;
  delete [] output_data;
  delete [] block_sum;
  delete [] cpu_output;
  return 0;
}