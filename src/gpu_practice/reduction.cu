
#include <hip/hip_runtime.h>
#include<cstdio>
#include<cstdlib>
#include<random>

using namespace std;

const int threads_per_block = 1024;

//将当前block利用共享内存进行reduction
__global__ void reduce(float* input, float* output){
  //共享内存申请
  __shared__ float shrared[threads_per_block];
  //坐标
  int id = threadIdx.x + blockDim.x * blockIdx.x;
  int tid = threadIdx.x;
  //从global memory读取数据到共享内存
  shrared[tid] = input[id];
  __syncthreads();

  //归约
  for(int s = blockDim.x / 2; s > 0; s >>= 1){
    if(tid < s){
      shrared[tid] += shrared[tid + s];
    }
    __syncthreads();
  }

  //将首元素填充到output;
  output[blockIdx.x] = shrared[0];
}

void reduction(float* data, float* result, float* part_result, int data_size){
  //计算需要多少block
  int num_blocks = (data_size + threads_per_block -1) / threads_per_block;
  //调用归约函数
  reduce<<<num_blocks,threads_per_block>>>(data, part_result);
  //现在得到part_result为每个block的和,归约该数组
  int final_step_num_threads = num_blocks;
  reduce<<<1,final_step_num_threads>>>(part_result,result);
}

int main(int argc, char** argv){
  //生成输入数据,1GB的float
  const int data_size = 1 << 20;
  float data[data_size];
  default_random_engine random;
  uniform_real_distribution<float> range(-2.0,2.0);
  float result = 0.0;
  for(int i = 0; i < data_size; i++){
    data[i] = range(random);
    result += data[i];
  }
  printf("数据完成");
  
  //gpu part
  float* dev_input;
  float* dev_output;
  float* dev_part;

  //分配内存
  hipMalloc((void**)&dev_input, data_size*sizeof(float));
  hipMalloc((void**)&dev_output, sizeof(float));
  int num_blocks = (data_size + threads_per_block -1) / threads_per_block;
  hipMalloc((void**)&dev_part, num_blocks*sizeof(float));

  //拷贝数据到设备
  hipMemcpy(dev_input, data, data_size*sizeof(float), hipMemcpyHostToDevice);
  //计算
  reduction(dev_input,dev_output,dev_part,data_size);
  //拷贝结果
  float dev_result = 0.0;
  hipMemcpy(&dev_result, dev_output, sizeof(float), hipMemcpyDeviceToHost);
  printf("cpu calculate:  %f,  cuda result:  %f", result, dev_result);

  hipFree(dev_input);
  hipFree(dev_output);
  hipFree(dev_part);
  return 0;
}